
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipError_t err;
  err = hipMallocManaged(&a, size);
  
  if ( err != hipSuccess){
      printf("Error: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }

  init(a, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();
  
  hipError_t syncErr = hipGetLastError();
  if ( syncErr != hipSuccess){
      printf("Error: %s\n", hipGetErrorString(syncErr));
  }
  
  hipError_t asyncErr = hipDeviceSynchronize();
   if ( asyncErr != hipSuccess){
      printf("Error: %s\n", hipGetErrorString(asyncErr));
  }

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
