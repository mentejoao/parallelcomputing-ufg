
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
    int globalIdx = threadIdx.x + blockIdx.x*blockDim.x;
    printf("This is iteration number %d\n", globalIdx);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  loop<<<2, 5>>>();
  hipDeviceSynchronize();
}
