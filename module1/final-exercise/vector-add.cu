#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

/* For adding error handling in locations where CUDA code might otherwise silently fail. /*
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}
// Augment the addVectorsInto definition so that it is a CUDA kernel
__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{

  /* Refactor the body of addVectorsInto: it will be launched inside of a single thread, and only needs to do one thread's worth of work on the input vectors. Be certain the thread will never try to access elements outside the range of the input vectors, and take care to note whether or not the thread needs to do work on more than one element of the input vectors. */
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  /* Update memory allocations, and memory freeing to reflect that the 3 vectors a, b, and result need to be accessed by host and device code. */
  checkCuda( hipMallocManaged(&a, size) );
  checkCuda( hipMallocManaged(&b, size) );
  checkCuda( hipMallocManaged(&c, size) );

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  /* Choose and utilize a working execution configuration so that addVectorsInto launches as a CUDA kernel. */
  threadsPerBlock = 256;
  numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  
  // Error Handling
  checkCuda( hipGetLastError() );
  checkCuda( hipDeviceSynchronize() );

  checkElementsAre(7, c, N);
  // Error Handling
  checkCuda( hipFree(a) );
  checkCuda( hipFree(b) );
  checkCuda( hipFree(c) );
}
